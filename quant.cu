#include "hip/hip_runtime.h"
template <typename scalar_t>
__global__ void E4M3_IEEE_Kernel(const scalar_t* __restrict__ in,
                                 scalar_t* __restrict__ out,
                                 const int size,
                                 const scalar_t in_scale,
                                 bool block_norm,
                                 int mbits,     // 8
                                 int exp_bits,  // 4
                                 int rmode) {
    int non_mant_bits = exp_bits + 1; /* exponent + sign */
    int lshift = 10 - (mbits - non_mant_bits);

    unsigned short rne_mask = 0;   /* round to nearest even mask */
    unsigned short rnaz_mask = 0;  /* round to nearest away from zero mask */
    unsigned short rntz_mask = 0;  /* round to nearest towards zero mask */
    unsigned short sr_mask = 0;    /* stochastic rounding mask */
    unsigned short rpinf_mask = 0; /* round to +INF */
    unsigned short rminf_mask = 0; /* round to -INF */

    if (rmode == ROUND_RNE)
        rne_mask = 1;
    if (rmode == ROUND_RNAZ)
        rnaz_mask = 1;
    if (rmode == ROUND_RNTZ)
        rntz_mask = 1;
    if (rmode == ROUND_STOCHASTIC)
        sr_mask = 1;
    if (rmode == ROUND_PINF)
        rpinf_mask = 1;
    if (rmode == ROUND_NINF)
        rminf_mask = 1;

    unsigned short mask_mant = (unsigned short)(0xFFFF << lshift);
    unsigned short grs_bitmask = 0x007F;
    unsigned short rne_tie = 0x00C0;

    extern __shared__ float sdata[];
    float scale = in_scale;

    if (block_norm == true) {
        absmax_block(in, sdata, size);
        __float_t f;
        f.f = sdata[0];
        f.u = (f.u & 0x7F800000);
        scale = 2 * f.f;
        scale = 2 * f.f;
        scale /= 8.0;
    }
    float scale_reciprocal = 1.0 / scale;

    for (int gid = (blockIdx.x * blockDim.x) + threadIdx.x; gid < size;
         gid += blockDim.x * gridDim.x) {
        __half_t h;
        float inval = in[gid] * scale;

        h.f = __anyfloat2half_rn(inval);
        short exp_h = (short)((h.u & 0x7C00) >> 10) - 15;
        short sign_h = (h.u & 0x8000);
        short mantissa_h = (h.u & 0x03FF);

        unsigned short can_round = ((h.u & 0x7FFF) < 0x4B80) ? 1 : 0;
        unsigned short is_normal =
            (((h.u & 0x7C00) <= 0x7800) && ((h.u & 0x7C00) >= 0x0400)) ? 1 : 0;
        unsigned short is_denorm = ((h.u & 0x7C00) == 0x0) ? 1 : 0;
        unsigned short is_naninf = ((h.u & 0x7C00) == 0x7C00) ? 1 : 0;

        int dshift = 0;

        if (exp_h > 7) {
            /* Hardware : saturate +/-INF */
            mantissa_h = 0;
            exp_h = 16;
            is_naninf = 1;
        } else if (exp_h < -9) {
            exp_h = -15;
            mantissa_h = 0;
        } else if (exp_h < -6) {
            dshift = (-6 - exp_h);
            /* handle denormals */
            mantissa_h = mantissa_h >> dshift;
            mantissa_h <<= dshift;
        }
        /* nearest rounding masks */
        unsigned short rnmask = (mantissa_h & grs_bitmask);
        unsigned short rnmask_tie = (mantissa_h & rne_tie);

        if (is_naninf == 0) {
            if (sr_mask) {
                /* stochastic with 16 seeds */
                int seed_index = (gid / 16);
                unsigned short rand =
                    (unsigned short)_rand_xorshft128plus_with_seed(sptr[(seed_index % 16)]);
                /* apply stochastic rounding before truncation if sr_mask is enabled */
                mantissa_h += can_round * is_normal * (rand & 0x7F);
                /* stochastic round:  denormals --> rne rounding */
                mantissa_h += can_round * is_denorm *
                              (((rnmask > 0x0040) || (rnmask_tie == rne_tie)) << lshift);
            } else {
                /* round to nearest even, if rne_mask is enabled */
                mantissa_h += can_round * rne_mask *
                              (((rnmask > 0x0040) || (rnmask_tie == rne_tie)) << lshift);
                /* round to nearest away from zero, if rnaz_mask is enabled */
                mantissa_h += can_round * rnaz_mask * ((rnmask >= 0x0040) << lshift);
                /* round to nearest towards zero, if rntz_mask is enabled */
                mantissa_h += can_round * rntz_mask * ((rnmask > 0x0040) << lshift);
                /* round to +INF, if rpinf_mask is enabled */
                mantissa_h += can_round * rpinf_mask * (h.f > 0) * ((rnmask >= 0x0040) << lshift);
                /* round to -INF, if rminf_mask is enabled */
                mantissa_h += can_round * rminf_mask * (h.f < 0) * ((rnmask >= 0x0040) << lshift);
            }
        }
        /* truncation */
        mantissa_h &= mask_mant;
        mantissa_h += ((exp_h + 15) << 10);
        mantissa_h |= sign_h;
        h.u = mantissa_h;
        __half2anyfloat(h.f * scale_reciprocal, &out[gid]);
    }
}